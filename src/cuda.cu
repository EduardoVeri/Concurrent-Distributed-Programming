// diff_eq_optimized.cu

#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <math.h>
#include <hip/hip_runtime.h>

#define CUDA_CHECK(err) \
    do { \
        hipError_t errr = (err); \
        if (errr != hipSuccess) { \
            fprintf(stderr, "CUDA Error: %s (err_num=%d) at %s:%d\n", \
            hipGetErrorString(errr), errr, __FILE__, __LINE__); \
            exit(errr); \
        } \
    } while (0)

// Structure to hold diffusion equation parameters
typedef struct {
    int N;
    double D;
    double DELTA_T;
    double DELTA_X;
} DiffEqArgs;

// Utility function to create and initialize a matrix
double** create_matrix_and_init(int N) {
    double **matrix = (double**)malloc(N * sizeof(double*));
    for (int i = 0; i < N; i++) {
        matrix[i] = (double*)calloc(N, sizeof(double));
    }
    // Initialize center
    matrix[N/2][N/2] = 1.0f;
    return matrix;
}

// Utility function to create a matrix
double** create_matrix(int N) {
    double **matrix = (double**)malloc(N * sizeof(double*));
    for (int i = 0; i < N; i++) {
        matrix[i] = (double*)calloc(N, sizeof(double));
    }
    return matrix;
}

// Utility function to free a matrix
void free_matrix(double **matrix, int N) {
    for (int i = 0; i < N; i++) {
        free(matrix[i]);
    }
    free(matrix);
}

// Optimized kernel combining compute and reduction
__global__ void compute_and_diff_kernel(const double *C, double *C_new, double *block_sums, int N, double D, double DELTA_T, double DELTA_X) {
    // Define shared memory for reduction
    extern __shared__ double sdata[];

    // Calculate global indices
    int i = blockIdx.y * blockDim.y + threadIdx.y;
    int j = blockIdx.x * blockDim.x + threadIdx.x;

    // Linear index for 1D representation
    int idx = i * N + j;

    // Initialize difference value
    double diff_val = 0.0f;

    // Perform computation if within bounds
    if (i > 0 && i < N-1 && j > 0 && j < N-1) {
        double center = C[idx];
        double up = C[(i-1)*N + j];
        double down = C[(i+1)*N + j];
        double left = C[i*N + (j-1)];
        double right = C[i*N + (j+1)];

        // Compute new value using diffusion equation
        double new_val = center + D * DELTA_T * ((up + down + left + right - 4.0f * center) / (DELTA_X * DELTA_X));
        C_new[idx] = new_val;

        // Compute absolute difference
        diff_val = fabsf(new_val - center);
    }

    // Load difference value into shared memory
    int tid = threadIdx.y * blockDim.x + threadIdx.x;
    sdata[tid] = diff_val;
    __syncthreads();

    // Perform reduction in shared memory
    // Reduce within a block
    for (unsigned int s = (blockDim.x * blockDim.y) / 2; s > 32; s >>= 1) {
        if (tid < s) {
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }

    // Unroll last warp
    if (tid < 32) {
        volatile double* vsmem = sdata;
        vsmem[tid] += vsmem[tid + 32];
        vsmem[tid] += vsmem[tid + 16];
        vsmem[tid] += vsmem[tid + 8];
        vsmem[tid] += vsmem[tid + 4];
        vsmem[tid] += vsmem[tid + 2];
        vsmem[tid] += vsmem[tid + 1];
    }

    // Write the block's partial sum to global memory
    if (tid == 0) {
        block_sums[blockIdx.y * gridDim.x + blockIdx.x] = sdata[0];
    }
}

// Host function to perform optimized diffusion equation computation
double cuda_diff_eq(double **C_host, double **C_new_host, DiffEqArgs *args, int T) {
    int N = args->N;
    double D = args->D;
    double DELTA_T = args->DELTA_T;
    double DELTA_X = args->DELTA_X;

    size_t size = N * N * sizeof(double);

    // Flatten host arrays
    double *C_flat = (double*)malloc(size);
    double *C_new_flat = (double*)malloc(size);
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            C_flat[i*N + j] = C_host[i][j];
            C_new_flat[i*N + j] = C_new_host[i][j];
        }
    }

    // Allocate device memory
    double *d_C, *d_C_new;
    CUDA_CHECK(hipMalloc((void**)&d_C, size));
    CUDA_CHECK(hipMalloc((void**)&d_C_new, size));
    CUDA_CHECK(hipMemcpy(d_C, C_flat, size, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(d_C_new, C_new_flat, size, hipMemcpyHostToDevice));

    // Define block and grid dimensions
    dim3 blockDim(16, 16); // Reduced block size for better occupancy
    dim3 gridDim((N + blockDim.x - 1)/blockDim.x, (N + blockDim.y - 1)/blockDim.y);

    int num_blocks = gridDim.x * gridDim.y;
    double *d_block_sums;
    CUDA_CHECK(hipMalloc((void**)&d_block_sums, num_blocks * sizeof(double)));

    // Allocate host memory for block sums
    double *h_block_sums = (double*)malloc(num_blocks * sizeof(double));

    size_t smem_size = blockDim.x * blockDim.y * sizeof(double);

    // Create CUDA stream for overlapping (optional)
    hipStream_t stream;
    CUDA_CHECK(hipStreamCreate(&stream));

    // Initialize pointers for pointer swapping
    double *current = d_C;
    double *next = d_C_new;

    // Main loop
    for (int t = 0; t < T; t++) {
        // Launch combined compute and reduction kernel
        compute_and_diff_kernel<<<gridDim, blockDim, smem_size, stream>>>(current, next, d_block_sums, N, D, DELTA_T, DELTA_X);
        CUDA_CHECK(hipGetLastError());

        // Every 100 steps, copy block sums to host and compute difmedio
        if (t % 100 == 0) {
            // Synchronize to ensure kernel has finished
            CUDA_CHECK(hipStreamSynchronize(stream));

            // Copy block sums to host
            CUDA_CHECK(hipMemcpyAsync(h_block_sums, d_block_sums, num_blocks * sizeof(double), hipMemcpyDeviceToHost, stream));
            CUDA_CHECK(hipStreamSynchronize(stream));

            // Compute total difference on host
            double total_diff = 0.0f;
            for (int i = 0; i < num_blocks; i++) {
                total_diff += h_block_sums[i];
            }
            double difmedio = total_diff / ((N-2)*(N-2));
            printf("Iteration %d - Difference = %g\n", t, difmedio);
        }

        // Swap pointers for next iteration
        double *temp = current;
        current = next;
        next = temp;
    }

    // Ensure all kernels have finished
    CUDA_CHECK(hipStreamSynchronize(stream));

    // Copy final data back to host
    CUDA_CHECK(hipMemcpy(C_flat, current, size, hipMemcpyDeviceToHost));

    // Reconstruct host matrix
    for (int i = 0; i < N; i++) {
        for (int j = 0; j < N; j++) {
            C_host[i][j] = C_flat[i*N + j];
        }
    }

    double final_val = C_host[N/2][N/2];

    // Cleanup
    free(C_flat);
    free(C_new_flat);
    free(h_block_sums);
    CUDA_CHECK(hipFree(d_C));
    CUDA_CHECK(hipFree(d_C_new));
    CUDA_CHECK(hipFree(d_block_sums));
    CUDA_CHECK(hipStreamDestroy(stream));

    return final_val;
}

int main(int argc, char *argv[]) {
    struct timeval start, end, start_parallel, end_parallel;
    gettimeofday(&start, NULL);

    if (argc != 7) {
        printf("Usage: %s <N> <T> <D> <DELTA_T> <DELTA_X> <NUM_THREADS>\n", argv[0]);
        return 1;
    }

    int N = atoi(argv[1]);
    int T = atoi(argv[2]);
    double D = atof(argv[3]);
    double DELTA_T = atof(argv[4]);
    double DELTA_X = atof(argv[5]);
    int NUM_THREADS = atoi(argv[6]); // Not used in CUDA

    // Initialize matrices
    double **C = create_matrix_and_init(N);
    double **C_new = create_matrix(N);

    DiffEqArgs args = {N, D, DELTA_T, DELTA_X};
    gettimeofday(&start_parallel, NULL);

    // Perform diffusion computation
    double final_val = cuda_diff_eq(C, C_new, &args, T);

    gettimeofday(&end_parallel, NULL);
    printf("Final concentration at center: %f\n", final_val);

    // Optionally save the matrix to a file
    // Implement salvar_matriz if needed

    free_matrix(C, N);
    free_matrix(C_new, N);

    gettimeofday(&end, NULL);
    double total_time = ((end.tv_sec * 1000000 + end.tv_usec) - (start.tv_sec * 1000000 + start.tv_usec)) / 1000.0;
    double parallel_time = ((end_parallel.tv_sec * 1000000 + end_parallel.tv_usec) - (start_parallel.tv_sec * 1000000 + start_parallel.tv_usec)) / 1000.0;
    double sequential_time = total_time - parallel_time;

    printf("Total Time: %lf ms\n", total_time);
    printf("Parallel Time (CUDA): %lf ms\n", parallel_time);
    printf("Sequential Time: %lf ms\n", sequential_time);

    return 0;
}
